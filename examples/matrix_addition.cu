
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

__device__ float add(const float x, const float y)
{
    return x + y;
}

__global__ void matrixAdd(float *A, float *B, float*C, const int nx, const int ny)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int id = ix + iy * nx;

    if (ix < nx && iy < ny)
    {
        C[id] = add(A[id], B[id]);
    }
}

void initialData(float *addr, int elementCount)
{
    std::mt19937 gen(std::random_device{}());
    std::uniform_real_distribution<float> dist(0.0f, 10.0f);

    for (int i = 0; i < elementCount; i++)
    {
        addr[i] = dist(gen);
    }
}

int main()
{
    // Get GPU devices count, and set GPU device
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess || deviceCount == 0)
    {
        printf("No GPU device found!\n");
        exit(-1);
    }
    else
    {
        printf("The number of GPU devices is %d.\n", deviceCount);
    }

    int rank = 0;
    error = hipSetDevice(rank);

    if (error != hipSuccess)
    {
        printf("Fail to set GPU device %d.\n", rank);
        exit(-1);
    }
    else
    {
        printf("Use GPU device %d.\n", rank);
    }

    // Allocate memory for host and device, and initilize
    int nx = 16;
    int ny = 8;
    int elementCount = nx * ny;
    size_t bytesCount = elementCount * sizeof(float);

    float *fpHost_A, *fpHost_B, *fpHost_C;
    fpHost_A = (float *)malloc(bytesCount);
    fpHost_B = (float *)malloc(bytesCount);
    fpHost_C = (float *)malloc(bytesCount);

    if (fpHost_A != NULL && fpHost_B != NULL && fpHost_C != NULL)
    {
        memset(fpHost_A, 0, bytesCount);
        memset(fpHost_B, 0, bytesCount);
        memset(fpHost_C, 0, bytesCount);
    }
    else
    {
        printf("Fail to allocate host memory!\n");
        free(fpHost_A);
        free(fpHost_B);
        free(fpHost_C);
        exit(-1);
    }

    float *fpDevice_A, *fpDevice_B, *fpDevice_C;
    hipMalloc(&fpDevice_A, bytesCount);
    hipMalloc(&fpDevice_B, bytesCount);
    hipMalloc(&fpDevice_C, bytesCount);

    if (fpDevice_A != NULL && fpDevice_B != NULL && fpDevice_C != NULL)
    {
        hipMemset(fpDevice_A, 0, bytesCount);
        hipMemset(fpDevice_B, 0, bytesCount);
        hipMemset(fpDevice_C, 0, bytesCount);
    }
    else
    {
        printf("Fail to allocate device memory!\n");
        free(fpDevice_A);
        free(fpDevice_B);
        free(fpDevice_C);
        exit(-1);
    }

    // Set values to host memory
    srand(666);
    initialData(fpHost_A, elementCount);
    initialData(fpHost_B, elementCount);

    // Copy data from host to device
    hipMemcpy(fpDevice_A, fpHost_A, bytesCount, hipMemcpyHostToDevice);
    hipMemcpy(fpDevice_B, fpHost_B, bytesCount, hipMemcpyHostToDevice);
    hipMemcpy(fpDevice_C, fpHost_C, bytesCount, hipMemcpyHostToDevice);

    // Launch kernel function to do matrix addition on device
    dim3 blockSize(4, 4);
    dim3 gridSize((nx + blockSize.x - 1) / blockSize.x, (ny + blockSize.y - 1) / blockSize.y);

    matrixAdd<<<gridSize, blockSize>>>(fpDevice_A, fpDevice_B, fpDevice_C, nx, ny);

    // Copy data from device to host
    // cudaDeviceSynchronize(); // This is not necessary because cudaMemcpy has implicate sync
    hipMemcpy(fpHost_C, fpDevice_C, bytesCount, hipMemcpyDeviceToHost);

    // Check the results
    for (int i = 0; i < 10; i++)
    {
        printf("idx=%d, A[idx]=%.1f, B[idx]=%.1f, C[idx]=%.1f\n", i, fpHost_A[i], fpHost_B[i], fpHost_C[i]);
    }

    // Free host and device memory
    free(fpHost_A);
    free(fpHost_B);
    free(fpHost_C);
    hipFree(fpDevice_A);
    hipFree(fpDevice_B);
    hipFree(fpDevice_C);

    hipDeviceReset();
    return 0;
}